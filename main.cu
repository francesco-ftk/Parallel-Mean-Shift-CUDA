#include <iostream>
#include <iostream>
#include <chrono>
#include "hip/hip_runtime.h"

#include "cpp_sources/ppm_io.cpp"
//#include "cpp_sources/matrix_meanshift.cpp"
//#include "cpp_sources/soa_meanshift.cpp"
#include "cpp_sources/rgb_pixels.cpp"

#include "cuda_sources/matrix_meanshift_cuda.cu"

#define INPUT_PATH "../img/image_bigger.ppm"
#define OUTPUT_PATH "../img/image_bigger_out_cuda.ppm"
#define ITERATIONS 1
#define BANDWIDTH 0.4

/* ----- TIMINGS ------------------------------
 * 100x100 image, Windows, 12 cores, 18 threads
 * 	 Matrix sequential: 3609ms
 * 	 Matrix OpenMP:		1029ms
 * 	   Speedup: 		3.5
 *   SoA sequential:	3834ms
 * 	 SoA OpenMP:		1060ms
 * 	   Speedup: 		3.6
 *
 * 100x100 image, Linux, 8 cores, 12 threads
 * 	 Matrix sequential:	2461ms
 * 	 Matrix OpenMP:		998ms
 * 	   Speedup:			2.5
 *   SoA sequential:	2711ms
 * 	 SoA OpenMP:		726ms
 * 	   Speedup:			3.7
 *
 * Averaged on 10 iterations
 * --------------------------------------------
 */

// todo: cluster in the HSV space
// todo: cluster in the L*U*V* space
// todo: kernel multiplication
// todo: parallelize using Cuda

using namespace std;
using namespace chrono;

int main()
{
	// open the ppm image
	PPM ppm;
	if (ppm.read(INPUT_PATH) != 0)
	{
		cout << "ERROR: failed to open the image";
		return -1;
	}
	int width = ppm.getW();
	int height = ppm.getH();
	int nOfPixels = width * height;
	uint8_t* inputBuffer = ppm.getImageHandler();

	// MATRIX MEANSHIFT START //

	// create the matrices
	int rgbPixelSize = RgbPixels::COLOR_SPACE_DIMENSION;
	int rgbxySpaceSize = RgbPixels::SPACE_DIMENSION;
	int rgbMaxValue = RgbPixels::MAX_VALUE;
	float* pixels = new float[nOfPixels * rgbxySpaceSize];
	float* modes = new float[nOfPixels * rgbxySpaceSize];

	// initialize the pixel data
	for (int i = 0; i < nOfPixels; ++i)
	{
		pixels[i * rgbxySpaceSize]     = (float) inputBuffer[i * rgbPixelSize]     / rgbMaxValue; // R
		pixels[i * rgbxySpaceSize + 1] = (float) inputBuffer[i * rgbPixelSize + 1] / rgbMaxValue; // G
		pixels[i * rgbxySpaceSize + 2] = (float) inputBuffer[i * rgbPixelSize + 2] / rgbMaxValue; // B
		pixels[i * rgbxySpaceSize + 3] = (float) ((i) % width) / (width - 1);					  // X
		pixels[i * rgbxySpaceSize + 4] = (float) ((i) / width) / (height - 1);					  // Y
	}

	// create the index array
	int* clusters = new int[nOfPixels];

	// create the result variables
	int nOfClusters;
	float totalTime = 0;

	// function loop
	for (int i = 0; i < ITERATIONS; ++i)
	{
		printf("Calling the MeanShift function... (%d)\n", i);

		// time the function
		auto start_time = high_resolution_clock::now();
		nOfClusters = matrixMeanShiftCUDA(pixels, nOfPixels, BANDWIDTH, rgbxySpaceSize, modes, clusters, width, height);
		auto end_time = high_resolution_clock::now();

		totalTime += duration_cast<microseconds>(end_time - start_time).count() / 1000.f;
	}

	float averageTime = totalTime / ITERATIONS;

	// print the results
	printf("Matrix timings: (measured on %d iterations)\n", ITERATIONS);
	printf("  total:   %fms\n", totalTime);
	printf("  average: %fms\n", averageTime);
	printf("Number of clusters: %d\n", nOfClusters);

	// MATRIX MEANSHIFT END //

	printf("\n");

	// SOA MEANSHIFT START //
/*
	// create the structures of arrays
	RgbPixels soaPixels;
	RgbPixels soaModes;
	soaPixels.create(width, height);
	soaModes.create(width, height);

	// initialize the pixel data
	soaPixels.load(inputBuffer);

	// create the index array
	//int clusters[nOfPixels];

	// create the result variables
	//int nOfClusters;
	totalTime = 0;

	// function loop
	for (int i = 0; i < ITERATIONS; ++i)
	{
		printf("Calling the MeanShift function... (%d)\n", i);

		// time the function
		auto start_time = high_resolution_clock::now();
		nOfClusters = soaMeanShift(soaPixels, nOfPixels, BANDWIDTH, soaModes, clusters);
		auto end_time = high_resolution_clock::now();

		totalTime += duration_cast<microseconds>(end_time - start_time).count() / 1000.f;
	}

	averageTime = totalTime / ITERATIONS;

	// print the results
	printf("SoA timings: (measured on %d iterations)\n", ITERATIONS);
	printf("  total:   %fms\n", totalTime);
	printf("  average: %fms\n", averageTime);
	printf("Number of clusters: %d\n", nOfClusters);

	// create the output image buffer
	rgbPixelSize = RgbPixels::COLOR_SPACE_DIMENSION;
	rgbMaxValue = RgbPixels::MAX_VALUE;
	uint8_t outputBuffer[nOfPixels * rgbPixelSize];
	for(int i = 0; i < nOfPixels; ++i)
	{
		outputBuffer[i * rgbPixelSize]     = (uint8_t) (soaModes.r[clusters[i]] * rgbMaxValue); // R
		outputBuffer[i * rgbPixelSize + 1] = (uint8_t) (soaModes.g[clusters[i]] * rgbMaxValue); // G
		outputBuffer[i * rgbPixelSize + 2] = (uint8_t) (soaModes.b[clusters[i]] * rgbMaxValue); // B
	}

	// free the memory
	soaPixels.destroy();
	soaModes.destroy();
*/
	// SOA MEANSHIFT END //

	// create the output image buffer
	rgbPixelSize = RgbPixels::COLOR_SPACE_DIMENSION;
	rgbMaxValue = RgbPixels::MAX_VALUE;
	uint8_t* outputBuffer = new uint8_t[nOfPixels * rgbPixelSize];
    for (int i = 0; i < nOfPixels; ++i)
	{
		outputBuffer[i * rgbPixelSize]	   = (uint8_t) (modes[clusters[i] * rgbxySpaceSize]     * rgbMaxValue); // R
		outputBuffer[i * rgbPixelSize + 1] = (uint8_t) (modes[clusters[i] * rgbxySpaceSize + 1] * rgbMaxValue); // G
		outputBuffer[i * rgbPixelSize + 2] = (uint8_t) (modes[clusters[i] * rgbxySpaceSize + 2] * rgbMaxValue); // B
	}

	ppm.load(outputBuffer, height, width, ppm.getMax(), ppm.getMagic());

	// write the output ppm image
	if (ppm.write(OUTPUT_PATH) != 0)
	{
		cout << "ERROR: failed to write the image";
		return -1;
	}

	delete[] pixels;
	delete[] modes;
	delete[] clusters;
	delete[] outputBuffer;

	return 0;
}

