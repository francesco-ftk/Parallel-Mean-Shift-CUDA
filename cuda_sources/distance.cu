
#include <hip/hip_runtime.h>
#include <cmath>

// Euclidean distance function
__device__ float l2Distance_cuda(float* row1, float* row2, size_t size) {
    float distance = 0;
    for (int i = 0; i < size; ++i) {
        distance += std::pow(row1[i] - row2[i], 2);
    }
    return sqrt(distance);
}

__device__ float l2SquaredDistance_cuda(float* row1, float* row2, size_t size) {
	float squaredDistance = 0;
	for (int i = 0; i < size; ++i) {
		squaredDistance += std::pow(row1[i] - row2[i], 2);
	}
	return squaredDistance;
}
