#pragma once
#include <hip/hip_runtime.h>


#include <iostream>
#include <string>

#define CUDA_CHECK_RETURN(value) checkCudaError(__FILE__,__LINE__, #value, value)

void checkCudaError(const char *file, unsigned line, const char *statement, hipError_t err) {
	if (err == hipSuccess) {
		return;
	}

	std::cerr << statement << " returned " << hipGetErrorString(err) <<
			  "(" << hipGetErrorName(err) << ") at " << file << ":" << std::to_string(line) << std::endl;
	exit (1);
}